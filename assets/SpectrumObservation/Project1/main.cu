#include "hip/hip_runtime.h"
/*
* Usage:
*	<this program> <data file> <config file>
*
* Binary Data Format must be real or complex number data such as
*	- <Re><Re><Re>...
*	- <Re><Im><Re><Im>...
*
*/

#include <cstdio>
#include <iostream>
#include <fstream>
#include <string>
#include <ctime>
#include "Functions.h"
#include "FileInformation.h"
#include "Config.h"
#include "BinData.h"
#include "Stats.h"

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <boost/filesystem.hpp>

#pragma comment(lib, "cudart")
#pragma comment(lib, "cufft")

#define STR_LEN_MAX 2048
#define CL

#ifdef CL
	#include <Windows.h>
	#define PLOT "pgnuplot"
#elif defined(GCC)
	#include <unistd.h>
	#define PLOT "gnuplot"
#endif

using std::cout;
using std::endl;
using std::string;
using boost::filesystem::path;

typedef struct {
	string xmin, xmax;
	string ymin, ymax;
	string zmin, zmax;
} plot_range;

void stft (hipfftComplex *h_idata, const int fft_window_width, const int fft_batch, bool inverse);
void normalize_data (hipfftComplex *partial_data, const int num_partial_data, const ta::Stats &stats);
void spectrogram (float **spectral_amp, float **spectral_phase, const int num_time_index, const int num_freq_index, const hipfftComplex *partial_data);
void mid_band_characteristics (float *band_amp, const int fft_batch, const int fft_window_width, float **spectral_amp);
void band_characteristics (float *band_amp, const int num_partition, const int fft_window_width);
void save_band_characteristics (const float *band_amp, const int fft_window_width, const bool flag_complex_data, ta::FileInformation &fileinfo, const path dest_dir);

void create_spectrogram_image (float **spectral_amp, float **spectral_phase, const int fft_batch, const int fft_window_width,  const int data_range_begin_pt, const int data_range_end_pt, const path dest_dir, ta::FileInformation &fileinfo, ta::Config &config);
int gnuplot(const char *data_filepath_char, const plot_range range, ta::Config &config);
//void time_series_integration_complex (int integration_point, ta::BinData &data, int num_entire_data_pt, ta::FileInformation &fileinfo, ta::Config &config, const path dest_dir)


int main(int argc, char *argv[])
{
	bool flag_create_spectrogram_image = false;

	try{
		// Input File
		if(argc != 3){
			printf("Usage:\n\t%s <data file> <confiig file>\n", argv[0]);
			throw "Invalid Arguments";
		}
		ta::FileInformation fileinfo (argv[1], '_');

		// Observation Parameters
		ta::Config config (argv[2]);
		const bool   flag_complex_data   = true; //config.isComplexData(); 
		const int    datum_bit_length_B  = config.getDatumBitLength_B(); // byte
		const double sampling_freq_Hz    = config.getSamplingFrequency_Hz(); // Hz, sample/sec
		const double sampling_interval_s = config.getSamplingInterval_s(); // sec

		const float  pulsar_p0_s         = config.getPulsarP0_s(); 
		const float  pulsar_w50_s        = config.getPulsarW50_s(); 


		int tmp_num_entire_data_pt = fileinfo.getFilesize_B() / datum_bit_length_B;
		if (flag_complex_data == true){
			tmp_num_entire_data_pt /= 2;
		}
		const int num_entire_data_pt = tmp_num_entire_data_pt;

		// Data Handling Parameters
		// �f�[�^�t�@�C���͑�e�ʂȂ̂ŁA�K���ȃf�[�^�T�C�Y�ɕ������ĉ�͂���B
		int tmp_num_partial_data = static_cast<int>(config.getPulsarP0_pt() * 5); // �����f�[�^���ɏ��Ȃ��Ƃ�1�̃p���X������悤�ɁA�����f�[�^�̓_����ݒ�B
		if (tmp_num_partial_data > num_entire_data_pt){ // �������S�f�[�^��������ɖ����Ȃ��ꍇ�A�����f�[�^����S�f�[�^���ɐݒ�B
			tmp_num_partial_data = num_entire_data_pt;
		}
		const int   fft_window_width       = config.getFFTWindowWidth();
		const int   fft_batch              = std::floor(static_cast<float>(tmp_num_partial_data) / fft_window_width);
		const int   num_partial_data       = fft_window_width * fft_batch;
		const int   num_partition          = std::floor( static_cast<float>(num_entire_data_pt) / num_partial_data);
		const int   skipped_data_size_pts  = num_entire_data_pt - num_partial_data * num_partition;

		// Display Parameters
		printf ("\nPulsar Information\n");
		printf ("- Period      P0      = %f ms\n", pulsar_p0_s  * 1000);
		printf ("- Pulse Width W50     = %f ms\n", pulsar_w50_s * 1000);
		printf ("\nData Information\n");
		printf ("- Sampling frequency  = %f MHz\n", sampling_freq_Hz / 1E+6);
		printf ("- Sampling interval   = %f ns\n",  sampling_interval_s * 1E+9);
		printf ("\nAnalysis Information\n");
		printf ("- FFT Window Width    = %d pt = %f ms\n", fft_window_width, fft_window_width * sampling_interval_s * 1000);
		printf ("- FFT Time resolution = %f ms\n", config.getFFTTimeResolution_s() * 1000);
		printf ("- FFT Freq resolution = %f kHz\n", config.getFFTFreqResolution_Hz() / 1000);
		if (config.getFFTTimeResolution_s() > pulsar_w50_s){
			printf("\n### CAUTION ###\nFFT Time Resolution > Pulse Width.\nThis analysis can not resolve the pulse profile.\n");
		}
		printf ("- Partial Data Size   = %f ms = %d pt = %d MB (on RAM)\n", sampling_interval_s * num_partial_data * 1000, num_partial_data, sizeof(float)*num_partial_data/static_cast<int>(1E+6));
		printf ("- Entire  Data Size   = %f ms = %d pt\n",                  sampling_interval_s * num_entire_data_pt * 1000, num_entire_data_pt);
		printf ("- Skipped Data Size   = %f ms = %d pt\n",                  sampling_interval_s * skipped_data_size_pts * 1000, skipped_data_size_pts);
		printf ("- Number of Analysis Partitions = %d\n", num_partition);

		printf ("\nInitiate the process?");
		if (!ta::stdin_yes_or_no()) {
			throw "Task Terminated";
		}

		printf ("\nAnalyzing...\n");

		// Data
		ta::BinData data;
		if (flag_complex_data == true) {
			clock_t t0 = clock();
			data.load_binary_as_double (fileinfo.getFilePath().string(), num_entire_data_pt, datum_bit_length_B);
			clock_t t1 = clock();
			ta::print_elapse ("Data Loading", t0, t1);
			
		} else {
			throw "not complex data";
		}

		// Create Directory
		const path dest_dir = path (config.getOutputDirectory()) / fileinfo.getFileStemSubstr();
		boost::filesystem::create_directories (dest_dir);
		boost::filesystem::current_path (dest_dir);
    
    //1s�ϕ��̓ǂݏo�� 
    //time_series_integration_complex (20000000, data, num_entire_data_pt, fileinfo, config, dest_dir) 

		// Initialize Parameters
		hipfftComplex *h_idata   = new hipfftComplex[num_partial_data];
		float **spectral_amp    = new float*[fft_batch]; // Power Spectral Density, psd[time][freq]
		float **spectral_phase  = new float*[fft_batch];
		for(int t=0; t<fft_batch; t++){
			spectral_amp[t]   = new float[fft_window_width];
			spectral_phase[t] = new float[fft_window_width];
		}
		for(int i=0; i<num_partial_data; i++){
			h_idata[i].x = 0.0;
			h_idata[i].y = 0.0;
		}

		float *gain_amp   = new float[fft_window_width];
		float *gain_phase = new float[fft_window_width];
		for(int f=0; f<fft_window_width; f++){
			gain_amp[f] = 0;
			gain_phase[f] = 0;
		}
		ta::Stats stats; // Statistic parameters including mean & variance values

		//
		// MAIN
		//
		for (int pos = 0; pos < num_partition; pos++) {
			const int partial_data_begin_pt = pos * num_partial_data;

			// Extract patial data
			if (flag_complex_data == true) {
				data.extract_binary_data_xy (h_idata, partial_data_begin_pt, num_partial_data);
			} else {
				throw "not complex data";
			}

			// Confirm Data For Debug
			if (pos == 0) {
				std::ofstream fout_test ( (dest_dir / (fileinfo.getFileStem().string() + ".txt")).string() );
				for (int i = 0; i < 100; i++) {
					fout_test << h_idata[i].x << "\t" << h_idata[i].y << "\n";
				}
				fout_test.close();
			}

			// Normalize the data using the mean & variance of the first partial data block (pos = 0)
			if (pos == 0) {
				stats.calcParams (h_idata, num_partial_data); // Create a Stats instance, calculating the mean & variance of the data.
			} 
			normalize_data (h_idata, num_partial_data, stats);

			// Forward FFT
			stft (h_idata, fft_window_width, fft_batch, false);

			// Create a spectrogram: spectral_amp[t][f], spectral_phase[t][f]
			clock_t spec0 = clock();
			spectrogram (spectral_amp, spectral_phase, fft_batch, fft_window_width, h_idata);
			clock_t spec1 = clock();
			ta::print_elapse ("Spectrogram Creating", spec0, spec1);

			// Derive a band characteristics gain_amp[f] from spectral_amp[t][f]
			mid_band_characteristics (gain_amp, fft_batch, fft_window_width, spectral_amp);
			
			if(pos == 0){
				//ta::saveSpectrumOfComplexData("gain amplitude at first data block.2d", gain_amp, fft_window_width);
				//ta::saveSpectrumOfComplexData("gain phase at first data block.2d", gain_phase, fft_window_width);
			}
			
			
			// Create a spectrogram image
			if(flag_create_spectrogram_image == true){
				create_spectrogram_image (spectral_amp, spectral_phase, fft_batch, fft_window_width, partial_data_begin_pt, partial_data_begin_pt + num_partial_data, dest_dir, fileinfo, config);
			}
		} // Next pos		
		
		// Band Characteristics
		band_characteristics (gain_amp, num_partition, fft_window_width);
		save_band_characteristics (gain_amp, fft_window_width, flag_complex_data, fileinfo, dest_dir);
		
/*
		//
		// Calibrate Data
		//

		float *calib_gain_amp = new float [fft_window_width];
		float *calib_gain_phase = new float [fft_window_width];
		for(int f=0; f<fft_window_width; f++){
			calib_gain_amp[f] = 0;
			calib_gain_phase[f] = 0;
		}

		for(int pos=0; pos<num_partition; pos++){

			// Load Data
			if(complex_data == true){
				loop_break_flag = loadBinaryDataComplex(partial_data_re, partial_data_im, num_partial_data, fileinfo.getFilePath().string().c_str(),  pos * datum_bit_length_B * num_partial_data, datum_bit_length_B);
			}else{
				loop_break_flag = loadBinaryData(partial_data_re, num_partial_data, fileinfo.getFilePath().string().c_str(),  pos * datum_bit_length_B * num_partial_data, datum_bit_length_B);
			}
			if(loop_break_flag != 0){ // Need to break also here.
				break;
			}

			

			// Normalize data using the mean & variance of the first data block (pos = 0)
			if(pos == 0){
				mean_re  = ta::mean(partial_data_re, num_partial_data);
				mean_im  = ta::mean(partial_data_im, num_partial_data);
				stdev_re = std::sqrt(ta::unbiased_variance(partial_data_re, num_partial_data, mean_re));
				stdev_im = std::sqrt(ta::unbiased_variance(partial_data_im, num_partial_data, mean_im));
			}	
			for(int i=0; i<num_partial_data; i++){
				partial_data_re[i] = (partial_data_re[i] - mean_re) / stdev_re;
				partial_data_im[i] = (partial_data_im[i] - mean_im) / stdev_im;
			}

			
			// Create input data on CPU
			// h_idata = input data on host CPU
			for(int i=0; i<num_partial_data; i++){
				h_idata[i].x = partial_data_re[i];
				h_idata[i].y = partial_data_im[i];
			}

			// Forward FFT
			STFT(h_idata, fft_window_width, fft_batch, false);

			// Create a spectrogram
			//h_idata[0].x = 0; h_idata[0].y = 0; // Remove the direct current (DC) component
			int index = 0; 
			const float root_N = std::sqrt(fft_window_width);
			for(int t=0; t<fft_batch; t++){
				for(int f=0; f<fft_window_width; f++){
					fft_re[f] = h_idata[index].x / root_N; 
					fft_im[f] = h_idata[index].y / root_N; // This assignment is only for readability.

					// Remove gain
					fft_re[f] /= gain_amp[f];
					fft_im[f] /= gain_amp[f];
					spectral_amp[t][f]   = std::hypot(fft_re[f], fft_im[f]);
					spectral_phase[t][f] = std::atan2(fft_im[f], fft_re[f]);

					h_idata[index].x = fft_re[f];
					h_idata[index].y = fft_im[f];
					index++;
				}
			}

			// Inverse FFT
			STFT(h_idata, fft_window_width, fft_batch, true);

			// Save Data
			
			if(pos < 5){
			unsigned int *test = new unsigned int [num_partial_data];
			for (int i=0; i<num_partial_data; i++){
				//test[i] = static_cast<unsigned int>(h_idata[i].x);
				//printf("(float %f, uint %u) ", h_idata[i].x, test[i]);
				fprintf(ofp_calib, "%.3f\t%.3f\n", h_idata[i].x / root_N, h_idata[i].y / root_N);
			}
			//fwrite(test, sizeof(float), num_partial_data, ofp_calib);
			delete [] test;
			}
			

			// Derive a band characteristics, gain_psd[f]
			for(int f=0; f<fft_window_width; f++){
				for(int t=0; t<fft_batch; t++){
					calib_gain_amp[f]   += spectral_amp[t][f] / fft_batch;
					calib_gain_phase[f] += spectral_phase[t][f] / fft_batch;
				}
				// Do not plase the division by fft_batch here.
			}
		} // Next pos	
		
		// Band Characteristics
		for(int f=0; f<fft_window_width; f++){
			calib_gain_amp[f]   /= num_partition;
			calib_gain_phase[f] /= num_partition;
		}
		calib_gain_amp[0] = 0;
		ta::smoothUsingBPF(calib_gain_amp, fft_window_width, 0, std::floor(static_cast<float>(fft_window_width)/2.0));
		if(complex_data == true){
			ta::saveSpectrumOfComplexData("calib_gain_amp.2d", calib_gain_amp, fft_window_width);		
			ta::saveSpectrumOfComplexData("calib_gain_phase.2d", calib_gain_phase, fft_window_width);
		}else{
			ta::saveData1D("calib_gain_amp.1d", calib_gain_amp, fft_window_width);
			ta::saveData1D("calib_gain_phase.1d", calib_gain_phase, fft_window_width);
		}
*/

		// Delete
		delete [] h_idata;
		for(int t=0; t<fft_batch; t++){
			delete [] spectral_amp[t];
			delete [] spectral_phase[t];
		}
		delete [] spectral_amp;
		delete [] spectral_phase;


		delete [] gain_amp;
		delete [] gain_phase;
		//delete [] calib_gain_amp;
		//delete [] calib_gain_phase;

		return 0;
	}
	catch (const char *err) {
		fprintf(stderr, "%s\n", err);
		system ("pause");
		return -1;
	}
	catch (const string err) {
		fprintf(stderr, "%s\n", err.c_str());
		system ("pause");
		return -1;
	}
}

void stft (hipfftComplex *h_idata, const int fft_window_width, const int fft_batch, bool inverse)
{
	const int NX = fft_window_width * fft_batch;

	// Transfer the data from CPU to GPU
	// d_idata = input data on device GPU
	hipfftComplex *d_idata;
	hipMalloc( (void**)&d_idata, sizeof(hipfftComplex) * NX);
	hipMemcpy(d_idata, h_idata,  sizeof(hipfftComplex) * NX, hipMemcpyHostToDevice);
			
	// Create an FFT plan
	hipfftHandle plan;
	hipfftPlan1d(&plan, fft_window_width, HIPFFT_C2C, fft_batch);

	// Execute the FFT
	if(inverse == false){
		hipfftExecC2C(plan, d_idata, d_idata, HIPFFT_FORWARD);
	}else{
		hipfftExecC2C(plan, d_idata, d_idata, HIPFFT_BACKWARD);
	}

	// Transfer data from GPU to CPU 
	hipMemcpy(h_idata, d_idata, sizeof(hipfftComplex) * NX, hipMemcpyDeviceToHost);

	// Erase the data on GPU
	hipFree(d_idata);
	hipfftDestroy(plan);
}



int gnuplot(const char *data_filepath_char, const plot_range range, ta::Config &config)
{
	// Gnuplot cannot recognize '\' in a file path, and therefore '\' must be repalced to '/'.
	string data_filepath   = data_filepath_char; ta::strReplace(data_filepath, "\\", "/");
	string config_filepath = data_filepath + ".plt";

	// Create a config file
	FILE *ofp = fopen(config_filepath.c_str(), "w");
	fprintf(ofp, "reset\n");
	fprintf(ofp, "set pm3d map corners2color max\n");
	fprintf(ofp, "set palette gray\n");
	fprintf(ofp, "set lmargin 0\n");
	fprintf(ofp, "set rmargin 0\n");
	fprintf(ofp, "set tmargin 0\n");
	fprintf(ofp, "set bmargin 0\n");
	
	fprintf(ofp, "set title \"STFT from #%s to #%s; Time resolution %f ms; Freq resolution %f kHz\"\n", range.xmin.c_str(), range.xmax.c_str(), config.getFFTTimeResolution_s() * 1E+3, config.getFFTFreqResolution_Hz() / 1000);
	fprintf(ofp, "set xlabel \"Time [%f ns]\"\n", config.getSamplingInterval_s() * 1E+9);
	fprintf(ofp, "set ylabel \"Frequency [%f kHz]\"\n", config.getFFTFreqResolution_Hz() / 1000);

	//fprintf(ofp, "set xrange [%d:%d]\n", 0, xend - xstart);
	fprintf(ofp, "set xrange [%s:%s]\n", range.xmin.c_str(), range.xmax.c_str());
	fprintf(ofp, "set yrange [%s:%s]\n", range.ymin.c_str(), range.ymax.c_str());
	fprintf(ofp, "set zrange [%s:%s]\n", range.zmin.c_str(), range.zmax.c_str()); // When data ~ N(0,1), spectral power ~ Exp(1/2). p-value = 28 for a significance level of 1E-6.
	fprintf(ofp, "set ytics 128\n");

	fprintf(ofp, "set terminal png font \"Verdana\" 10 size 4800,3000\n"); // font = Verdana or Serif; Consolas looks dirty.
	fprintf(ofp, "set output \"%s.png\"\n", data_filepath.c_str());
	fprintf(ofp, "splot \"%s\" using 1:2:3 notitle\n", data_filepath.c_str());
	fprintf(ofp, "# If skipping some data, use an splot option 'every 5:5'.\n");
	//fprintf(ofp, "splot \"%s\" using ($1 - %d):2:3 notitle\n", input_data_file_path, xstart);

	fprintf(ofp, "\n"); // Important \n. If it does not exist, gnuplot may not progress.
	fclose(ofp);

	// Load the config file and plot the data
	FILE *gp = _popen(PLOT, "w");
	fprintf(gp, "load \"%s\"\n", config_filepath.c_str());
	fprintf(gp, "exit\n");
	_pclose(gp);
	
	return 0;
}

void normalize_data (hipfftComplex *partial_data, const int num_partial_data, const ta::Stats &stats)
{
	const float mean_x = stats.getMean_x ();
	const float mean_y = stats.getMean_y ();
	const float inverse_stdev_x = 1.0 / stats.getStdev_x ();
	const float inverse_stdev_y = 1.0 / stats.getStdev_y ();
	for (int i=0; i<num_partial_data; i++) {
		partial_data[i].x = (partial_data[i].x - mean_x) * inverse_stdev_x;
		partial_data[i].y = (partial_data[i].y - mean_y) * inverse_stdev_y;
	}
}

void spectrogram (float **spectral_amp, float **spectral_phase, const int num_time_index, const int num_freq_index, const hipfftComplex *partial_data)
{
	int index = 0; 
	for (int t = 0; t < num_time_index; t++) {
		for (int f = 0; f < num_freq_index; f++) {
			spectral_amp[t][f]   = static_cast<float>(std::hypot (partial_data[index].x, partial_data[index].y));
			spectral_phase[t][f] = static_cast<float>(std::atan2 (partial_data[index].y, partial_data[index].x));
			index++;
		}
	}
}

void mid_band_characteristics (float *band_amp, const int fft_batch, const int fft_window_width, float **spectral_amp)
{
	// Do NOT initialize band_amp[f], becuuse the parameter has been and will be integrated.
	const float inverse_fft_batch = 1.0 / fft_batch;
	for (int f = 0; f < fft_window_width; f++) {
		for (int t = 0; t < fft_batch; t++) {
			band_amp[f] += spectral_amp[t][f] * inverse_fft_batch;
			//band_phase[f] += spectral_phase[t][f] * inverse_fft_batch; No practical meaning. The spectral phase is random noise.
		}
	}
}

void band_characteristics (float *band_amp, const int num_partition, const int fft_window_width)
{
	// Derive a band characteristics
	const float inverse_num_partition = 1.0 / static_cast<float>(num_partition);
	for (int f = 0; f < fft_window_width; f++){
		band_amp[f] *= inverse_num_partition;
	}

	// Smoothe the band characteristics
	ta::smoothUsingBPF (band_amp, fft_window_width, 0, std::floor(static_cast<float>(fft_window_width)/2.0));

	// Normalize data using the minimum value
	const float min_band_amp = ta::minimum (band_amp, fft_window_width);
	const float inverse_min_gain_amp = 1.0 / min_band_amp;
	for(int f=0; f<fft_window_width; f++){
		band_amp[f] *= inverse_min_gain_amp;
	}
	band_amp[0] = 0;
}

void save_band_characteristics (const float *band_amp, const int fft_window_width, const bool flag_complex_data, ta::FileInformation &fileinfo, const path dest_dir)
{
	if (flag_complex_data == true) {
		path fout = dest_dir / (fileinfo.getFileStem().string() + ".spectral_amplitude.2d");
		ta::saveSpectrumOfComplexData(fout.string().c_str(), band_amp, fft_window_width);	
	}
	else {
		path fout = dest_dir / (fileinfo.getFileStem().string() + ".spectral_amplitude.1d");
		ta::saveData1D(fout.string().c_str(), band_amp, fft_window_width);
	}
}

void create_spectrogram_image (float **spectral_amp, float **spectral_phase, const int fft_batch, const int fft_window_width,  const int data_range_begin_pt, const int data_range_end_pt, const path dest_dir, ta::FileInformation &fileinfo, ta::Config &config)
{
	char str[STR_LEN_MAX + 1];

	// Save the spectrogram as a text file
	const int data_range_begin_B = config.getDatumBitLength_B() * data_range_begin_pt;
	const int data_range_end_B   = config.getDatumBitLength_B() * data_range_end_pt;

	sprintf (str, "%s.%d-%dB.spectrogram.3d", fileinfo.getFileStem().string().c_str(), data_range_begin_B, data_range_end_B);
	const path output_specamp_filepath   = dest_dir / str; // Specify the destination directory and avoid depending on the current directory for Gnuplot configuration.
	FILE *ofp_spec = fopen(output_specamp_filepath.string().c_str(), "w"); if(ofp_spec == NULL){throw ta::messageFileOpenError(output_specamp_filepath.string());}

	sprintf (str, "%s.%d-%dB.phase.3d", fileinfo.getFileStem().string().c_str(), data_range_begin_B, data_range_end_B);
	const path output_specphase_filepath = dest_dir / str; // Specify the destination directory and avoid depending on the current directory for Gnuplot configuration.
	FILE *ofp_phase = fopen(output_specphase_filepath.string().c_str(), "w"); if(ofp_phase == NULL){throw ta::messageFileOpenError(output_specphase_filepath.string());}

	for(int t=0; t<fft_batch; t++){
		for(int f=0; f<fft_window_width; f++){
			fprintf(ofp_spec,  "%d\t%d\t%.3f\n", data_range_begin_pt + t * fft_window_width, f, spectral_amp[t][f] * spectral_amp[t][f]);
			fprintf(ofp_phase, "%d\t%d\t%.2f\n", data_range_begin_pt + t * fft_window_width, f, spectral_phase[t][f]);
		}
		fprintf(ofp_spec, "\n");
		fprintf(ofp_phase, "\n");
	}
	fclose(ofp_spec);
	fclose(ofp_phase);


	// Make an PNG image of the spectrogram
	plot_range range;
	range.xmin = data_range_begin_pt;
	range.xmax = data_range_end_pt;
	range.ymin = "0";
	range.ymax = fft_window_width;
	range.zmin = "0";
	range.zmax = "";
	gnuplot (output_specamp_filepath.string().c_str(), range, config);

	range.zmin = "- 4";
	range.zmax = "4";
	gnuplot (output_specphase_filepath.string().c_str(), range, config);
}

void time_series_integration_complex (const float integration_time_s, ta::FileInformation &fileinfo, ta::BinData &data, ta::Config &config, const path dest_dir)
{
	const int num_entire_data_pt = data.getNumData();
	const int integration_time_pt = integration_time_s / config.getSamplingInterval_s();
	const int num_integ_data_pt = num_entire_data_pt / integration_time_pt + 1; // Important +1
	const int num_residue_pt = num_entire_data_pt % integration_time_pt;

	hipfftComplex *data_xy = new hipfftComplex[integration_time_pt];
	float *tmp_data_x = new float[integration_time_pt];
	float *tmp_data_y = new float[integration_time_pt];
	float *integ_data_x = new float[num_integ_data_pt];
	float *integ_data_y = new float[num_integ_data_pt];

	for (int pos = 0; pos < num_integ_data_pt; pos++) {
		data.extract_binary_data_xy (data_xy, pos * integration_time_pt, integration_time_pt);
		for (int i = 0; i < integration_time_pt; i++) {
			tmp_data_x[i] = data_xy[i].x;
			tmp_data_y[i] = data_xy[i].y;
		}

		if (pos < num_integ_data_pt - 1) {
			integ_data_x[pos] = ta::mean (tmp_data_x, integration_time_pt);
			integ_data_y[pos] = ta::mean (tmp_data_y, integration_time_pt);
		} else {
			integ_data_x[pos] = ta::mean (tmp_data_x, num_residue_pt);
			integ_data_y[pos] = ta::mean (tmp_data_y, num_residue_pt);
		}
	}

//	path filepath_re = 
//	path filepath_im = 
//	ta::saveData1D (filepath_re.string().c_str(), integ_data_x, num_integ_data_pt);
//	ta::saveData1D (filepath_re.string().c_str(), integ_data_x, num_integ_data_pt);
	//ta::saveData2D (filepath_re.string().c_str(), integ_data_x, num_integ_data_pt, integration_time_pt);
	//ta::saveData2D (filepath_im.string().c_str(), integ_data_y, num_integ_data_pt, integration_time_pt);


	
	delete [] data_xy;
	delete [] tmp_data_x;
	delete [] tmp_data_y;
	delete [] integ_data_x;
	delete [] integ_data_y;
}

/**
  �E�E���݂��߂΂��������ȁ[
  integration_point ���|�C���g�̍��v�l����邩(��b�ϕ��Ȃ�1/50^[-9]�Ȃ̂�20000000�|�C���g)
  data �o�C�i������ǂݎ�����f�[�^�z������炤�B�X�y�N�g���̑傫���̔z����󂯎�肽��(ta::BinData data.load_binary_as_double�ł�񂾃f�[�^)
  num_entire_data_pt �f�[�^�̑S�|�C���g��
  tmp_num_partial_data �����f�[�^�̃|�C���g��(����Ȃ������A�Ă�����Ȃ���)
  fileinfo file�̃p�X���Ƃ��󂯎��
  config config�̓ǂݎ��
  dest_dir �ۑ���̃f�B���N�g�����ȁH
  complex_flag�̓I���Ɖ���
**/

/**
void time_series_integration_complex (int integration_point, ta::BinData &data, int num_entire_data_pt, ta::FileInformation &fileinfo, ta::Config &config, const path dest_dir)
{
  
  int tmp_num_partial_data = 100000000 //�K���ɃZ�b�g(�K�v�Ȃ��B�J��integrate_window_width�y�����������悳��)
  const int   integrate_window_width  = integration_point;
  const int   integrate_batch = std::floor(static_cast<float>(tmp_num_partial_data) / integrate_window_width);
  const int   num_partial_data       = integrate_window_width * integrate_batch;
  const int   num_partition          = std::floor( static_cast<float>(num_entire_data_pt) / num_partial_data);
  const int   skipped_data_size_pts  = num_entire_data_pt - num_partial_data * num_partition;

  //1�b�ϕ��̂��߂ɗp��
  float *integrate_re = new float[integrate_batch * num_partition];
  float *integrate_im = new float[integrate_batch * num_partition];
  for(int f=0; f<integrate_batch * num_partition; f++){
    integrate_re[f] = 0;
    integrate_im[f] = 0;
  }
  
  float *h_idata   = new float[num_partial_data];
  for(int i=0; i<num_partial_data; i++){
    h_idata[i].x = 0.0;
    h_idata[i].y = 0.0;
  }
  
  //
  // MAIN
  //
  
  for (int pos = 0; pos < num_partition; pos++) {
    const int partial_data_begin_pt = pos * num_partial_data;
    //hi_data�̍쐬
    // �����̏�����re��im�̂��ꂼ��̋��x���|�C���g���ɓ������Ηǂ�
    if (flag_complex_data == true) {
      const{

        float *cudata = h_idata
        const int extraction_first_point = partial_data_begin_pt
        const int extraction_width = num_partial_data
          
        if( !load_d_executed ){return -1;}
        
        if (extraction_first_point < 0 || extraction_width < 0 || extraction_width > num_data){
          throw "Exception: �֐� Data::extractData() �̈����s��";
        }
        if (extraction_first_point + extraction_width> num_data){
          for(int i=0; i<extraction_width; i++){
            cudata[i].x = 0;
            cudata[i].y = 0;
          }
          for(int i=0; i<num_data - extraction_first_point; i++){
            //�����Ńf�[�^�̎��ԂƋ��x����߂�Ηǂ�����re, im�ŏo���Ă�?
            cudata[i].x = static_cast<float>(static_cast<unsigned>(data[extraction_first_point + i].x));
            cudata[i].y = static_cast<float>(static_cast<unsigned>(data[extraction_first_point + i].y));
          }
        }else{
          for(int i=0; i<extraction_width; i++){
            //�����Ńf�[�^�̎��ԂƋ��x����߂�Ηǂ�����re, im�ŏo���Ă�?
            cudata[i].x = static_cast<float>(static_cast<unsigned>(data[extraction_first_point + i].x)); // start ���� start+width-1 �|�C���g�̃f�[�^�𒊏o
            cudata[i].y = static_cast<float>(static_cast<unsigned>(data[extraction_first_point + i].y)); // start ���� start+width-1 �|�C���g�̃f�[�^�𒊏o
          }
        }
        return 0;
      }
    } else {
      throw "not complex data";
    }

    // Confirm Data For Debug
    if (pos == 0) {
      std::ofstream fout_test ( (dest_dir / (fileinfo.getFileStem().string() + ".txt")).string() );
      for (int i = 0; i < 100; i++) {
        fout_test << h_idata[i].x << "\t" << h_idata[i].y << "\n";
      }
      fout_test.close();
    }
    
    //��b�ϕ�
    for(j=0; j < integrate_batch ; j++){
      //������
      double re=0.0;
      double im=0.0;
      int integrate_first_point = j * integrate_window_width + pos * num_partial_data;
      
      for(i=0; i < integrate_window_width; i++)
      {
        re+=h_idata[i+integrate_first_point];
        im+=h_idata[i+integrate_first_point];
      }
      integrate_re[j + pos * integrate_batch] = re;
      integrate_im[j + pos * integrate_batch] = im;
      //�f�o�b�O
      printf("Intergral re : %lf, im : %lf\n", re , im);
    }

  } // Next pos		
	
	// Save the Integrate as a text file
	char str[STR_LEN_MAX + 1];

	sprintf (str, "%s.integrate.im", fileinfo.getFileStem().string().c_str());
	const path output_im_filepath   = dest_dir / str; // Specify the destination directory and avoid depending on the current directory for Gnuplot configuration.
	FILE *ofp_im = fopen(output_im_filepath.string().c_str(), "w"); if(ofp_im == NULL){throw ta::messageFileOpenError(output_im_filepath.string());}
	
	sprintf (str, "%s.integrate.re", fileinfo.getFileStem().string().c_str());
  const path output_re_filepath   = dest_dir / str; // Specify the destination directory and avoid depending on the current directory for Gnuplot configuration.
	FILE *ofp_re = fopen(output_re_filepath.string().c_str(), "w"); if(ofp_re == NULL){throw ta::messageFileOpenError(output_re_filepath.string());}

	for(int t=0; t<integrate_batch * num_partition; t++){
    fprintf(ofp_im,  "%f\n", integrate_im[t]);
    fprintf(ofp_re,  "%f\n", integrate_re[t]);
	}
	
  fclose(ofp_im);
	fclose(ofp_re);
  
  delete [] h_idata;
  delete [] integrate_re;
  delete [] integrate_im;
}
**/
