#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "../cuda_by_example/common/book.h"
#include <time.h>
#define imin(a,b) (a<b?a:b)

#define N 50000
#define M 10000
const int threadsPerBlock = 500;
const int blocksPerGrid =
            imin( 60000, (N+threadsPerBlock-1) / threadsPerBlock );

__global__ void add( float *a, float *b, float *c ) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < N)
      c[tid] = (a[tid] / b[tid] + 120)*a[tid]/b[tid] + (a[tid] / b[tid] + 120)*a[tid]/b[tid] + (a[tid] / b[tid] + 120)*a[tid]/b[tid] +  (a[tid] / b[tid] + 120)*a[tid]/b[tid] + (a[tid] / b[tid] + 120)*a[tid]/b[tid];
      c[tid] += (a[tid] / b[tid] + 120)*a[tid]/b[tid] + (a[tid] / b[tid] + 120)*a[tid]/b[tid] + (a[tid] / b[tid] + 120)*a[tid]/b[tid] +  (a[tid] / b[tid] + 120)*a[tid]/b[tid] + (a[tid] / b[tid] + 120)*a[tid]/b[tid];
      c[tid] += (a[tid] / b[tid] + 120)*a[tid]/b[tid] + (a[tid] / b[tid] + 120)*a[tid]/b[tid] + (a[tid] / b[tid] + 120)*a[tid]/b[tid] +  (a[tid] / b[tid] + 120)*a[tid]/b[tid] + (a[tid] / b[tid] + 120)*a[tid]/b[tid];
      c[tid] = c[tid]/N/N;
      tid += blockDim.x * gridDim.x;
}

int main( void ) {
    clock_t start, end;
    float a[N], b[N], c[N];
    int i,j;
    float *dev_a, *dev_b, *dev_c;
    start = clock();

    for(j=0; j<M;j++) {
      // allocate the memory on the GPU
      HANDLE_ERROR( hipMalloc( (void**)&dev_a, N * sizeof(float)) );
      HANDLE_ERROR( hipMalloc( (void**)&dev_b, N * sizeof(float)) );
      HANDLE_ERROR( hipMalloc( (void**)&dev_c, N * sizeof(float)) );
      
      // fill the arrays 'a' and 'b' on the CPU
      for (i=0; i<N; i++) {
        a[i] =  (i);
        b[i] =  (i+j);
        if (j==230){
          //printf( "%f / %f = \n", a[i], b[i] );
          //printf( "%f / %f = %f\n", a[i], b[i], c[i] );
        }
      }
      
      // copy the arrays 'a' and 'b' to the GPU
      HANDLE_ERROR( hipMemcpy( dev_a, a, N * sizeof(float),
                                hipMemcpyHostToDevice ) );
      HANDLE_ERROR( hipMemcpy( dev_b, b, N * sizeof(float),
                                hipMemcpyHostToDevice ) );

      //add<<<N,1>>>( dev_a, dev_b, dev_c );
      add<<<blocksPerGrid,threadsPerBlock>>>( dev_a, dev_b, dev_c );

      // copy the array 'c' back from the GPU to the CPU
      HANDLE_ERROR( hipMemcpy( c, dev_c,   N * sizeof(float),
                                hipMemcpyDeviceToHost ) );
      //if (j==230){
        for (int i=0; i<N; i++) {
          //printf( "%f + %f = %f\n", a[i], b[i], c[i] );
          //printf( "%f \n", c[i] );
        }
      //}
      // free the memory allocated on the GPU
      HANDLE_ERROR( hipFree( dev_a ) );
      HANDLE_ERROR( hipFree( dev_b ) );
      HANDLE_ERROR( hipFree( dev_c ) );
    }
    
    end = clock();
    printf("%.2f秒かかりました\n",(double)(end-start)/CLOCKS_PER_SEC);
    return 0;
}
